#include "hip/hip_runtime.h"
/*
#ifndef __HIPCC__
#define __HIPCC__
#endif
*/

#include <iostream>
#include <stdio.h>
#include <cmath>
#include <fstream>
#include <string>
#include <cstdlib>
#include <iterator>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_cooperative_groups.h>

#include "hip/hip_runtime.h"
#include ""

#include <algorithm>

#define SUBSEQLENGTH 10
#define NUMTHREADS 32
#define TSLENGTH 24125                      // Lunghezza txt
//#define TSLENGTH 2299                     // Lunghezza csv
#define NUMBLOCKS (TSLENGTH + NUMTHREADS - 1) / NUMTHREADS            // Calcolo del numero ottimale di blocchi

__constant__ double primo_vettore_confronto[SUBSEQLENGTH];            // Si crea il vettore della memoria constant e lo si riempe con una parte
                                                                      // del vettore timeSeries lungo SUBSEQLENGTH partendo dall'indice i

using namespace std;


__inline__ __device__ void warpReduceMin(double& val, int& idx) {

    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        double tmpVal = __shfl_down_sync(0xFFFFFFFF, val, offset, 32);
        int tmpIdx = __shfl_down_sync(0xFFFFFFFF, idx, offset, 32);

        if (tmpVal == val) {                                          // Se due valori di distanza sono uguai si salva la posizione pi� piccola
            if (tmpIdx < idx) {
                idx = tmpIdx;
            }
        }

        if (tmpVal < val) {
            if (!(tmpVal == 0)) {                                    // TODO verifica dei valori = 0. Alcuni sono corretti in quanto la distanza tra due sottosequenza � effettivamente 0
                val = tmpVal;
                idx = tmpIdx;
            }
        }
    }
}


__inline__ __device__ void blockReduceMin(double& val, int& idx, int currentThreads, int indexFirstSubsequence) {

    static __shared__ double values[32], indices[32];               // Shared mem for 32 partial mins
    int lane = threadIdx.x % warpSize;
    int wid = threadIdx.x / warpSize;

    warpReduceMin(val, idx);                                        // Each warp performs partial reduction

    if (lane == 0) {
        values[wid] = val;                                          // Write reduced value to shared memory
        indices[wid] = idx;                                         // Write reduced value to shared memory
    }

    __syncthreads();                                                // Wait for all partial reductions

    if (threadIdx.x < (blockDim.x + warpSize - 1) / warpSize && threadIdx.x <= currentThreads / warpSize) {     // Read from shared memory only if that warp existed
        val = values[lane];
        idx = indices[lane];
    }
    else {
        val = DBL_MAX;
        idx = 0;
    }

    if (wid == 0) {
        warpReduceMin(val, idx);                                    // Final reduce within first warp
    }
}


__global__ void sequencesDistance(int indexFirstSubsequence, double* dev_timeSeries, double* dev_blocksDistances, int* dev_blocksLocations) {

    extern __shared__ double cache[];                                    // La sua lunghezza � NUMTHREADS + SUBSEQLENGTH - 1   
    double sum = 0;
    double distanza = DBL_MAX;                                           // Variabile a cui � assegnata la distanza. I thread non coinvolti nel calcolo hanno questo valore come default
    double diff;
    double power;

    int first_arr_index = indexFirstSubsequence;                         // Indice della prima sottosequenza che verr� confrontata con tutte le altre
    int second_arr_index = blockIdx.x * blockDim.x + threadIdx.x;        // Indice della seconda sottosequenza che si confronta con la prima. L'id globale di ogni thread stabilisce il punto di partenza
    int altro_indice = blockIdx.x * blockDim.x + threadIdx.x;
    int indice_cache = threadIdx.x;


    while (indice_cache < (NUMTHREADS + SUBSEQLENGTH - 1) && altro_indice <= TSLENGTH) {       // Ogni thread carica nella shared uno o pi� elementi di timeSeries
        cache[indice_cache] = dev_timeSeries[altro_indice];
        indice_cache = indice_cache + NUMTHREADS;
        altro_indice = altro_indice + NUMTHREADS;
    }

    __syncthreads();                                                     // SYNC perch� non tutti possono aver gi� riempito la cache  


    if (abs(second_arr_index - first_arr_index) >= SUBSEQLENGTH && second_arr_index < TSLENGTH - SUBSEQLENGTH + 1) {    // Verifica di self-match e controllo che il secondo indice rientri nei valori consentiti
        for (int i = 0; i < SUBSEQLENGTH; i++) {
            diff = primo_vettore_confronto[i] - cache[threadIdx.x + i];
            power = diff * diff;
            sum = sum + power;
        }
        distanza = sqrt(sum);                                            // Da rimuovere nella versione finale, la radice si calcola alla fine
    }

    blockReduceMin(distanza, second_arr_index, NUMTHREADS, indexFirstSubsequence);

    if (threadIdx.x == 0) {
        dev_blocksDistances[blockIdx.x] = distanza;
        dev_blocksLocations[blockIdx.x] = second_arr_index;
    }
}


__global__ void finalReduction(int indexFirstSubsequence, int previousBlocks, int* dev_blocksLocations, double* dev_blocksDistances, int* dev_finalLocations, double* dev_finalDistances) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < previousBlocks) {
        double val = dev_blocksDistances[tid];
        int idx = dev_blocksLocations[tid];

        blockReduceMin(val, idx, previousBlocks, indexFirstSubsequence);

        if (tid == 0 && gridDim.x != 1) {           // Si utilizzano i vettori dev_blocksDistances e dev_blocksLocations per salvare i risultati delle riduzioni
            dev_blocksDistances[blockIdx.x] = val;  // ad ogni nuova iterazione finch� si utilizza pi� di un blocco 
            dev_blocksLocations[blockIdx.x] = idx;
        }

        if (tid == 0 && gridDim.x == 1) {               // Si scrive sui vettori finali solo quando la riduzione � arrivata ad utilizzare un solo blocco
            dev_finalDistances[indexFirstSubsequence] = val;
            dev_finalLocations[indexFirstSubsequence] = idx;
        }
    }
}


void compareSubsequences(double* dev_blocksDistances, int* dev_blocksLocations, double* dev_timeSeries, double* dev_finalDistances, int* dev_finalLocations) {

    int threads = 1024;

    for (int i = 0; i <= TSLENGTH - SUBSEQLENGTH; i++) {        // Outer loop

        int previousBlocks = NUMBLOCKS;
        int currentBlocks = (NUMBLOCKS + threads - 1) / threads;

        bool continueReduction = true;

        hipMemcpyToSymbol(HIP_SYMBOL(primo_vettore_confronto), &dev_timeSeries[i], SUBSEQLENGTH * sizeof(double), 0, hipMemcpyDeviceToDevice);  // Copia nella constant la sottosequenza all'i-esima posizione 
                                                                                                                                      // da confrontare con tutte le altre

        sequencesDistance << <NUMBLOCKS, NUMTHREADS, (NUMTHREADS + SUBSEQLENGTH - 1) * sizeof(double) >> > (i, dev_timeSeries, dev_blocksDistances, dev_blocksLocations);  // Kernel che esegue il calcolo delle distanze 
                                                                                                                                                                      // ed una prima riduzione

        while (continueReduction) {
            finalReduction << <currentBlocks, threads >> > (i, previousBlocks, dev_blocksLocations, dev_blocksDistances, dev_finalLocations, dev_finalDistances); // Riduce i risultati ottenuti dal kernel precedente

            if (currentBlocks == 1) {
                continueReduction = false;
            }

            previousBlocks = currentBlocks;
            currentBlocks = (currentBlocks + threads - 1) / threads;
        }
    }
    hipFree(primo_vettore_confronto);
}


//--------------------Lettura e scrittura file start--------------------------
void readFile(double* timeSeries, string fileName) {

    const char* c = fileName.c_str();
    double num = 0;
    int i = 0;
    ifstream readFile;
    readFile.open(c);

    if (!readFile.is_open()) {                                                       // Check to see that the file was opened correctly
        cerr << "There was a problem opening the input file!\n";
        exit(1);                                                                     // Exit or do additional error checking
    }

    while (readFile >> num) {                                                        // Keep storing values from the text file so long as data exists
        timeSeries[i] = double(num);
        i++;
    }

    readFile.close();
}


void scriviFile(double* distances, int* locations, string fileName) {

    FILE* fp;
    char nomeFile;
    //   nomeFile = "nnd.dat";

    fp = fopen("nnd.dat", "w+");
    for (int i = 0; i < TSLENGTH - SUBSEQLENGTH + 1; i++) {
        fprintf(fp, "%lf \n", distances[i]);
    }
    fclose(fp);

    //    nomeFile = "ngh"+fileName  +".dat";
    fp = fopen("loc.dat", "w+");
    for (int i = 0; i < TSLENGTH - SUBSEQLENGTH + 1; i++) {
        fprintf(fp, "%d \n ", locations[i] + 1);
    }
    fclose(fp);

    fp = fopen("total.dat", "w+");
    for (int i = 0; i < TSLENGTH - SUBSEQLENGTH + 1; i++) {
        fprintf(fp, "%lf \t %d \n", distances[i], locations[i] + 1);
    }
    fclose(fp);
}
//--------------------Lettura e scrittura file end-----------------------------


int main() {

    //string fileName = "ecg0606_1.csv";
    string fileName = "nprs44.txt";

    double* timeSeries;
    double* distances;
    int* locations;

    timeSeries = (double*)malloc(TSLENGTH * sizeof(double));
    distances = (double*)malloc(TSLENGTH * sizeof(double));
    locations = (int*)malloc(TSLENGTH * sizeof(int));

    fill_n(distances, TSLENGTH, DBL_MAX);

    cout << "File name: " << fileName << endl;
    cout << "File length: " << TSLENGTH << endl;
    cout << "Subsequence length: " << SUBSEQLENGTH << endl;
    cout << "Number of blocks: " << NUMBLOCKS << endl;
    cout << "Number of threads: " << NUMTHREADS << endl;

    readFile(timeSeries, fileName);

    double* dev_blocksDistances;                 // Vettore sulla memoria global dove ogni blocco salva il risultato della distanza che calcola
    int* dev_blocksLocations;                    // Vettore sulla memoria global dove ogni blocco salva l'indice della propria migliore distanza trovata 
    double* dev_finalDistances;
    int* dev_finalLocations;
    double* dev_timeSeries;

    hipMalloc((void**)&dev_blocksDistances, NUMBLOCKS * sizeof(double));
    hipMalloc((void**)&dev_blocksLocations, NUMBLOCKS * sizeof(int));
    hipMalloc((void**)&dev_timeSeries, TSLENGTH * sizeof(double));
    hipMalloc((void**)&dev_finalDistances, TSLENGTH * sizeof(double));
    hipMalloc((void**)&dev_finalLocations, TSLENGTH * sizeof(int));

    hipMemcpy(dev_timeSeries, timeSeries, TSLENGTH * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_finalDistances, distances, TSLENGTH * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_finalLocations, locations, TSLENGTH * sizeof(int), hipMemcpyHostToDevice);

    compareSubsequences(dev_blocksDistances, dev_blocksLocations, dev_timeSeries, dev_finalDistances, dev_finalLocations);     // Funzione che lancia il kernel

    hipMemcpy(locations, dev_finalLocations, TSLENGTH * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(distances, dev_finalDistances, TSLENGTH * sizeof(double), hipMemcpyDeviceToHost);

    scriviFile(distances, locations, fileName);

    free(timeSeries);
    free(distances);
    free(locations);

    hipFree(dev_blocksDistances);
    hipFree(dev_blocksLocations);
    hipFree(dev_timeSeries);
    hipFree(dev_finalDistances);
    hipFree(dev_finalLocations);

    return 0;
}
